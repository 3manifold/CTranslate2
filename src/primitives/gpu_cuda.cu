#include "hip/hip_runtime.h"
#include "ctranslate2/primitives/gpu_cuda.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/device_vector.h>

#include "ctranslate2/types.h"
#include "ctranslate2/cuda/utils.h"

namespace ctranslate2 {

  template <typename T, typename UnaryFunction>
  void unary_transform(const T* x, T* y, size_t size, UnaryFunction op) {
    thrust::transform(thrust::cuda::par.on(cuda::get_cuda_stream()), x, x + size, y, op);
  }

  template <typename T, typename BinaryFunction>
  void binary_transform(const T* a, const T* b, T* c, size_t size, BinaryFunction op) {
    thrust::transform(thrust::cuda::par.on(cuda::get_cuda_stream()), a, a + size, b, c, op);
  }


  template<>
  void* primitives<Device::CUDA>::alloc_data(size_t size) {
    void* data = nullptr;
    CUDA_CHECK(hipMalloc(&data, size));
    return data;
  }

  template<>
  void primitives<Device::CUDA>::free_data(void* data) {
    CUDA_CHECK(hipFree(data));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::fill(T* x, T a, size_t size) {
    thrust::fill_n(thrust::cuda::par.on(cuda::get_cuda_stream()), x, size, a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::copy(const T* x, T* y, size_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T),
                               hipMemcpyDeviceToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::sum(const T* array, size_t size) {
    return thrust::reduce(thrust::cuda::par.on(cuda::get_cuda_stream()), array, array + size);
  }

  template<>
  template <typename T>
  size_t primitives<Device::CUDA>::max_element(const T* array, size_t size) {
    const auto* max = thrust::max_element(thrust::cuda::par.on(cuda::get_cuda_stream()),
                                          array, array + size);
    return static_cast<size_t>(max - array);
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::max(const T* array, size_t size) {
    thrust::device_ptr<const T> array_ptr(array);
    return *thrust::max_element(thrust::cuda::par.on(cuda::get_cuda_stream()),
                                array_ptr, array_ptr + size);
  }

  template<>
  template <typename T, typename I>
  void primitives<Device::CUDA>::topk(const T* x, T* val, I* ind, size_t k, size_t size) {
    static thread_local T* keys = nullptr;
    static thread_local I* values = nullptr;
    static thread_local size_t alloc_size = 0;

    if (size > alloc_size) {
      CUDA_CHECK(hipFree(keys));
      CUDA_CHECK(hipMalloc(&keys, size * sizeof (T)));
      CUDA_CHECK(hipFree(values));
      CUDA_CHECK(hipMalloc(&values, size * sizeof (I)));
      alloc_size = size;
    }

    copy(x, keys, size);
    thrust::sequence(thrust::cuda::par.on(cuda::get_cuda_stream()), values, values + size);
    thrust::sort_by_key(thrust::cuda::par.on(cuda::get_cuda_stream()),
                        keys, keys + size, values, thrust::greater<T>());
    copy(keys, val, k);
    copy(values, ind, k);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(T a, const T* x, T* y, size_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 + a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(const T* a, const T* b, T* c, size_t size) {
    binary_transform(a, b, c, size, thrust::plus<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, size_t size) {
    binary_transform(a, b, c, size, thrust::minus<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(T a, const T* x, T* y, size_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 * a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, size_t size) {
    binary_transform(a, b, c, size, thrust::multiplies<T>());
  }

  struct relu_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return fmaxf(x, 0); }
  };

  template<>
  template<>
  void primitives<Device::CUDA>::relu(const float* x, float* y, size_t size) {
    unary_transform(x, y, size, relu_func());
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm(const float* a, const float* b,
                                      bool transpose_a, bool transpose_b,
                                      size_t m, size_t n, size_t k,
                                      float alpha, float beta,
                                      float* c) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasSgemm(cuda::get_cublas_handle(),
                             transb, transa,
                             n, m, k,
                             &alpha,
                             b, ldb,
                             a, lda,
                             &beta,
                             c, ldc));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm_batch(const float* a, const float* b,
                                            bool transpose_a, bool transpose_b,
                                            size_t batch_size,
                                            size_t m, size_t n, size_t k,
                                            float alpha, float beta,
                                            float* c) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    const float** a_array = new const float*[batch_size];
    const float** b_array = new const float*[batch_size];
    float** c_array = new float*[batch_size];

    for (size_t i = 0; i < batch_size; ++i) {
      a_array[i] = a + (i * m * k);
      b_array[i] = b + (i * k * n);
      c_array[i] = c + (i * m * n);
    }

    static thread_local const float** a_array_device = nullptr;
    static thread_local const float** b_array_device = nullptr;
    static thread_local float** c_array_device = nullptr;
    static thread_local size_t alloc_size = 0;

    const size_t array_size = batch_size * sizeof (float*);

    if (array_size > alloc_size) {
      CUDA_CHECK(hipFree(a_array_device));
      CUDA_CHECK(hipFree(b_array_device));
      CUDA_CHECK(hipFree(c_array_device));
      CUDA_CHECK(hipMalloc(&a_array_device, array_size));
      CUDA_CHECK(hipMalloc(&b_array_device, array_size));
      CUDA_CHECK(hipMalloc(&c_array_device, array_size));
      alloc_size = array_size;
    }

    cross_device_primitives<Device::CPU, Device::CUDA>::copy(a_array, a_array_device, batch_size);
    cross_device_primitives<Device::CPU, Device::CUDA>::copy(b_array, b_array_device, batch_size);
    cross_device_primitives<Device::CPU, Device::CUDA>::copy(c_array, c_array_device, batch_size);

    delete [] a_array;
    delete [] b_array;
    delete [] c_array;

    CUBLAS_CHECK(hipblasSgemmBatched(cuda::get_cublas_handle(),
                                    transb, transa,
                                    n, m, k,
                                    &alpha,
                                    b_array_device, ldb,
                                    a_array_device, lda,
                                    &beta,
                                    c_array_device, ldc,
                                    batch_size));
  }

  struct exp_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return expf(x); }
  };

  template<>
  template<>
  void primitives<Device::CUDA>::exp(const float* x, float* y, size_t size) {
    unary_transform(x, y, size, exp_func());
  }


  template<>
  template <typename T>
  void cross_device_primitives<Device::CPU, Device::CUDA>::copy(const T* x, T* y, size_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyHostToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  void cross_device_primitives<Device::CUDA, Device::CPU>::copy(const T* x, T* y, size_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyDeviceToHost, cuda::get_cuda_stream()));
  }

#define DECLARE_IMPL(T)                                                 \
  template void                                                         \
  primitives<Device::CUDA>::fill(T* x, T a, size_t size);               \
  template void                                                         \
  primitives<Device::CUDA>::copy<T>(const T* x, T* y, size_t size);     \
  template T                                                            \
  primitives<Device::CUDA>::sum(const T* array, size_t size);           \
  template size_t                                                       \
  primitives<Device::CUDA>::max_element(const T* array, size_t size);   \
  template T                                                            \
  primitives<Device::CUDA>::max(const T* array, size_t size);           \
  template void                                                         \
  primitives<Device::CUDA>::topk(const T* x, T* values, int* indices, size_t k, size_t size); \
  template void                                                         \
  primitives<Device::CUDA>::add(T a, const T* x, T* y, size_t size);    \
  template void                                                         \
  primitives<Device::CUDA>::add(const T* a, const T* b, T* c, size_t size); \
  template void                                                         \
  primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, size_t size); \
  template void                                                         \
  primitives<Device::CUDA>::mul(T a, const T* x, T* y, size_t size);    \
  template void                                                         \
  primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, size_t size); \
  template void                                                         \
  cross_device_primitives<Device::CPU, Device::CUDA>::copy<T>(const T*, T*, size_t); \
  template void                                                         \
  cross_device_primitives<Device::CUDA, Device::CPU>::copy<T>(const T*, T*, size_t);

  DECLARE_ALL_TYPES(DECLARE_IMPL)

}
