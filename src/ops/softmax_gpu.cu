#include "ctranslate2/ops/softmax.h"
#include "ctranslate2/ops/log_softmax.h"

#include "ctranslate2/cuda/utils.h"

namespace ctranslate2 {
  namespace ops {

    static void cudnn_softmax(const StorageView& input,
                              StorageView& output,
                              hipdnnSoftmaxAlgorithm_t algorithm) {
      static thread_local hipdnnTensorDescriptor_t tensor_desc;
      static thread_local bool tensor_desc_init = false;

      if (!tensor_desc_init) {
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensor_desc));
        tensor_desc_init = true;
      }

      size_t depth = input.dim(-1);
      size_t batch_size = input.size() / depth;
      CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensor_desc,
                                             HIPDNN_TENSOR_NCHW,
                                             HIPDNN_DATA_FLOAT,
                                             batch_size,
                                             depth,
                                             1 /* h */, 1 /* w */));

      float alpha = 1;
      float beta = 0;
      CUDNN_CHECK(hipdnnSoftmaxForward(cuda::get_cudnn_handle(),
                                      algorithm,
                                      HIPDNN_SOFTMAX_MODE_INSTANCE,
                                      &alpha,
                                      tensor_desc,
                                      input.data<float>(),
                                      &beta,
                                      tensor_desc,
                                      output.data<float>()));
    }

    template <Device D, typename T>
    void SoftMax::compute(const StorageView& input, StorageView& output) const {
      cudnn_softmax(input, output, HIPDNN_SOFTMAX_ACCURATE);
    }

    template <Device D, typename T>
    void LogSoftMax::compute(const StorageView& input, StorageView& output) const {
      cudnn_softmax(input, output, HIPDNN_SOFTMAX_LOG);
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    SoftMax::compute<Device::CUDA, T>(const StorageView& input,         \
                                      StorageView& output) const;       \
    template void                                                       \
    LogSoftMax::compute<Device::CUDA, T>(const StorageView& input,      \
                                         StorageView& output) const;

    DECLARE_IMPL(float)

  }
}
