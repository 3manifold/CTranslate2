#include "ctranslate2/ops/softmax.h"

#include "ctranslate2/cuda/utils.h"

namespace ctranslate2 {
  namespace ops {

    template <Device D, typename T>
    void SoftMax::compute(const StorageView& input, StorageView& output) const {
      static thread_local hipdnnTensorDescriptor_t tensor_desc;
      static thread_local bool tensor_desc_init = false;

      if (!tensor_desc_init) {
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensor_desc));
        tensor_desc_init = true;
      }

      size_t depth = input.dim(-1);
      size_t batch_size = input.size() / depth;
      CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensor_desc,
                                             HIPDNN_TENSOR_NCHW,
                                             HIPDNN_DATA_FLOAT,
                                             batch_size,
                                             depth,
                                             1 /* h */, 1 /* w */));

      T alpha = 1;
      T beta = 0;
      CUDNN_CHECK(hipdnnSoftmaxForward(cuda::get_cudnn_handle(),
                                      HIPDNN_SOFTMAX_ACCURATE,
                                      HIPDNN_SOFTMAX_MODE_INSTANCE,
                                      &alpha,
                                      tensor_desc,
                                      input.data<T>(),
                                      &beta,
                                      tensor_desc,
                                      output.data<T>()));
    }

#define DECLARE_IMPL(T)                                         \
    template void                                               \
    SoftMax::compute<Device::CUDA, T>(const StorageView& input, \
                                      StorageView& output) const;

    DECLARE_IMPL(float)

  }
}
