#include "hip/hip_runtime.h"
#include "ctranslate2/ops/topk.h"

#include <iostream>
#include <memory>

#include <NvInfer.h>

#include "ctranslate2/cuda/utils.h"

namespace ctranslate2 {
  namespace ops {

    class Logger : public nvinfer1::ILogger {
      void log(Severity severity, const char* msg) override {
        // suppress info-level messages
        if (severity != Severity::kINFO)
          std::cerr << msg << std::endl;
      }
    } g_logger;

    class Allocator : public nvinfer1::IGpuAllocator {
      void* allocate(uint64_t size, uint64_t, uint32_t) override {
        return primitives<Device::CUDA>::alloc_data(size);
      }

      void free(void* memory) override {
        primitives<Device::CUDA>::free_data(memory);
      }

    } g_allocator;

    class TopKLayer {
    public:
      TopKLayer(nvinfer1::IBuilder* builder, int k, int depth)
        : _depth(depth) {
        _network = builder->createNetwork();
        nvinfer1::Dims input_dim{1, {depth}, {nvinfer1::DimensionType::kCHANNEL}};
        nvinfer1::ITensor* input = _network->addInput("x", nvinfer1::DataType::kFLOAT, input_dim);
        nvinfer1::ITopKLayer* topk = _network->addTopK(*input, nvinfer1::TopKOperation::kMAX, k, 1);
        nvinfer1::ITensor* values_t = topk->getOutput(0);
        nvinfer1::ITensor* indices_t = topk->getOutput(1);
        _network->markOutput(*values_t);
        _network->markOutput(*indices_t);
        values_t->setName("values");
        indices_t->setName("indices");
        indices_t->setType(nvinfer1::DataType::kINT32);
        _engine = builder->buildCudaEngine(*_network);
        _execution_context = _engine->createExecutionContext();
      }

      ~TopKLayer() {
        _network->destroy();
        _engine->destroy();
      }

      nvinfer1::IExecutionContext* get_execution_context() {
        return _execution_context;
      }

      int depth() const {
        return _depth;
      }

    private:
      int _depth;
      nvinfer1::INetworkDefinition* _network;
      nvinfer1::ICudaEngine* _engine;
      nvinfer1::IExecutionContext* _execution_context;
    };

    static int max_batch_size = 512;

    template <Device D, typename DataType, typename IndexType>
    void TopK::compute(const StorageView& x,
                       StorageView& values,
                       StorageView& indices) const {
      static thread_local nvinfer1::IBuilder* builder = nullptr;
      if (!builder) {
        builder = nvinfer1::createInferBuilder(g_logger);
        builder->setMaxBatchSize(max_batch_size);
        builder->setMaxWorkspaceSize(1 << 30);
        builder->setGpuAllocator(&g_allocator);
      }

      int depth = x.dim(-1);
      int batch_size = x.size() / depth;

      if (batch_size > max_batch_size)
        throw std::runtime_error("Maximum batch size supported by the TopK layer is "
                                 + std::to_string(max_batch_size) + ", but got "
                                 + std::to_string(batch_size));

      static thread_local std::unique_ptr<TopKLayer> topk_layer;
      if (!topk_layer || topk_layer->depth() != depth)
        topk_layer.reset(new TopKLayer(builder, _k, depth));

      void* bindings[3] = {
        const_cast<DataType*>(x.data<DataType>()),
        values.data<DataType>(),
        indices.data<IndexType>()
      };

      topk_layer->get_execution_context()->enqueue(batch_size,
                                                   bindings,
                                                   cuda::get_cuda_stream(),
                                                   NULL);
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    TopK::compute<Device::CUDA, T, int32_t>(const StorageView& x,       \
                                            StorageView& values,        \
                                            StorageView& indices) const;

    DECLARE_IMPL(float)

  }
}
